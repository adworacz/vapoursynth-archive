#include "hip/hip_runtime.h"
//////////////////////////////////////////
// This file contains a simple invert
// filter that's commented to show
// the basics of the filter api.
// This file may make more sense when
// read from the bottom and up.

#include <stdlib.h>
#include "VapourSynth.h"
#include "VSHelper.h"
#include "VSCuda.h"

typedef struct {
    VSNodeRef *node;
    const VSVideoInfo *vi;
    int enabled;
} InvertData;

// This function is called immediately after vsapi->createFilter(). This is the only place where the video
// properties may be set. In this case we simply use the same as the input clip. You may pass an array
// of VSVideoInfo if the filter has more than one output, like rgb+alpha as two separate clips.
static void VS_CC invertInit(VSMap *in, VSMap *out, void **instanceData, VSNode *node, VSCore *core, const VSAPI *vsapi) {
    InvertData *d = (InvertData *) * instanceData;
    vsapi->setVideoInfo(d->vi, 1, node);
}

// This is the main function that gets called when a frame should be produced. It will, in most cases, get
// called several times to produce one frame. This state is being kept track of by the value of
// activationReason. The first call to produce a certain frame n is always arInitial. In this state
// you should request all the input frames you need. Always do it in ascending order to play nice with the
// upstream filters.
// Once all frames are ready, the the filter will be called with arAllFramesReady. It is now time to
// do the actual processing.
static const VSFrameRef *VS_CC invertGetFrame(int n, int activationReason, void **instanceData, void **frameData, VSFrameContext *frameCtx, VSCore *core, const VSAPI *vsapi) {
    InvertData *d = (InvertData *) * instanceData;

    if (activationReason == arInitial) {
        // Request the source frame on the first call
        vsapi->requestFrameFilter(n, d->node, frameCtx);
    } else if (activationReason == arAllFramesReady) {
        const VSFrameRef *src = vsapi->getFrameFilter(n, d->node, frameCtx);
        // The reason we query this on a per frame basis is because we want our filter
        // to accept clips with varying dimensions. If we reject such content using d->vi
        // would be better.
        const VSFormat *fi = d->vi->format;
        int height = vsapi->getFrameHeight(src, 0);
        int width = vsapi->getFrameWidth(src, 0);


        // When creating a new frame for output it is VERY EXTREMELY SUPER IMPORTANT to
        // supply the "dominant" source frame to copy properties from. Frame props
        // are an essential part of the filter chain and you should NEVER break it.
        VSFrameRef *dst = vsapi->newVideoFrame(fi, width, height, src, core);

        // It's processing loop time!
        // Loop over all the planes
        int plane;
        for (plane = 0; plane < fi->numPlanes; plane++) {
            const uint8_t *srcp = vsapi->getReadPtr(src, plane);
            int src_stride = vsapi->getStride(src, plane);
            uint8_t *dstp = vsapi->getWritePtr(dst, plane);
            int dst_stride = vsapi->getStride(dst, plane); // note that if a frame has the same dimensions and format, the stride is guaranteed to be the same. int dst_stride = src_stride would be fine too in this filter.
            // Since planes may be subsampled you have to query the height of them individually
            int h = vsapi->getFrameHeight(src, plane);
            int y;
            int w = vsapi->getFrameWidth(src, plane);
            int x;

            for (y = 0; y < h; y++) {
                for (x = 0; x < w; x++)
                    dstp[x] = ~srcp[x];

                dstp += dst_stride;
                srcp += src_stride;
            }
        }

        // Release the source frame
        vsapi->freeFrame(src);

        // A reference is consumed when it is returned, so saving the dst reference somewhere
        // and reusing it is not allowed.
        return dst;
    }

    return 0;
}

static __global__ void invertKernel(uint8_t *d_srcdata, uint8_t *d_dstdata, int width, int height) {
    const int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);

    if (ix > width || iy > height)
        return;

    d_dstdata[width * iy + ix] = ~d_srcdata[width * iy + ix];
}

static void invertWithCuda(VSFrameRef *src, VSFrameRef *dst, const VSFormat *fi, const VSAPI *vsapi){
    hipPitchedPtr d_srcp;
    hipPitchedPtr d_dstp;


    int deviceID = 0;
    hipDeviceProp_t deviceProp;
    CHECKCUDA(hipGetDeviceProperties(&deviceProp, deviceID));

    //CUDA Compute Capability < 2.0 only supports a maximum of 512 threads,
    //while CUDA Compute Capability >= 2.0 supports 1024 threads.
    int blockSize = (deviceProp.major < 2) ? 16 : 32;

    int plane;
    for (plane = 0; plane < fi->numPlanes; plane++) {
        int h = vsapi->getFrameHeight(src, plane);
        int w = vsapi->getFrameWidth(src, plane);

        const uint8_t *srcp = vsapi->getReadPtr(src, plane);
        uint8_t *dstp = vsapi->getWritePtr(dst, plane);
        int src_stride = vsapi->getStride(src, plane);
        int dst_stride = vsapi->getStride(dst, plane);

        if (plane > 0)
            h = h >> fi->subSamplingH; //Account for subsampling in non-luma planes

        //Allocate GPU memory for src frame and dst frame, then ship over src data.
        CHECKCUDA(hipMalloc3D(&d_srcp, make_hipExtent(w * fi->bytesPerSample, h, 1)));
        CHECKCUDA(hipMalloc3D(&d_dstp, make_hipExtent(w * fi->bytesPerSample, h, 1)));

        CHECKCUDA(hipMemcpy2D(d_srcp.ptr, d_srcp.pitch, srcp, src_stride, w * fi->bytesPerSample, h, hipMemcpyHostToDevice));

        //Do processing.
        dim3 threads(blockSize, blockSize);
        dim3 grid(ceil(w / threads.x), ceil(h / threads.y));

        invertKernel<<<grid, threads>>>((uint8_t *)d_srcp.ptr, (uint8_t *)d_dstp.ptr, w, h);

        //Free up GPU memory.
        CHECKCUDA(hipFree(d_srcp.ptr));
        CHECKCUDA(hipFree(d_dstp.ptr));
    }
}

// Free all allocated data on filter destruction
static void VS_CC invertFree(void *instanceData, VSCore *core, const VSAPI *vsapi) {
    InvertData *d = (InvertData *)instanceData;
    vsapi->freeNode(d->node);
    free(d);
}

// This function is responsible for validating arguments and creating a new filter
static void VS_CC invertCreate(const VSMap *in, VSMap *out, void *userData, VSCore *core, const VSAPI *vsapi) {
    InvertData d;
    InvertData *data;
    VSNodeRef *cref;
    int err;

    // Get a clip reference from the input arguments. This must be freed later.
    d.node = vsapi->propGetNode(in, "clip", 0, 0);
    d.vi = vsapi->getVideoInfo(d.node);

    // In this first version we only want to handle 8bit integer formats. Note that
    // vi->format can be 0 if the input clip can change format midstream.
    if (!isConstantFormat(d.vi) || d.vi->format->sampleType != stInteger || d.vi->format->bitsPerSample != 8) {
        vsapi->setError(out, "cuInvert: only constant format 8bit integer input supported");
        vsapi->freeNode(d.node);
        return;
    }

    // If a property read fails for some reason (index out of bounds/wrong type)
    // then err will have flags set to indicate why and 0 will be returned. This
    // can be very useful to know when having optional arguments. Since we have
    // strict checking because of what we wrote in the argument string, the only
    // reason this could fail is when the value wasn't set by the user.
    // And when it's not set we want it to default to enabled.
    d.enabled = !!vsapi->propGetInt(in, "enable", 0, &err);
    if (err)
        d.enabled = 1;

    // Let's pretend the only allowed values are 1 or 0...
    if (d.enabled < 0 || d.enabled > 1) {
        vsapi->setError(out, "cuInvert: enabled must be 0 or 1");
        vsapi->freeNode(d.node);
        return;
    }

    // I usually keep the filter data struct on the stack and don't allocate it
    // until all the input validation is done.
    data = (InvertData *)malloc(sizeof(d));
    *data = d;

    // Creates a new filter and returns a reference to it. Always pass on the in and out
    // arguments or unexpected things may happen. The name should be something that's
    // easy to connect to the filter, like its function name.
    // The three function pointers handle initialization, frame processing and filter destruction.
    // The filtermode is very important to get right as it controls how threading of the filter
    // is handled. In general you should only use fmParallel whenever possible. This is if you
    // need to modify no shared data at all when the filter is running.
    // For more complicated filters, fmParallelRequests is usually easier to achieve as it can
    // be prefetched in parallel but the actual processing is serialized.
    // The others can be considered special cases where fmSerial is useful to source filters and
    // fmUnordered is useful when a filter's state may change even when deciding which frames to
    // prefetch (such as a cache filter).
    // If your filter is really fast (such as a filter that only resorts frames) you should set the
    // nfNoCache flag to make the caching work smoother.
    vsapi->createFilter(in, out, "cuInvert", invertInit, invertGetFrame, invertFree, fmParallel, 0, data, core);
    return;
}

//////////////////////////////////////////
// Init

// This is the entry point that is called when a plugin is loaded. You are only supposed
// to call the two provided functions here.
// configFunc sets the id, namespace, and long name of the plugin (the last 3 arguments
// never need to be changed for a normal plugin).
//
// id: Needs to be a "reverse" url and unique among all plugins.
//   It is inspired by how android packages identify themselves.
//   If you don't own a domain then make one up that's related
//   to the plugin name.
//
// namespace: Should only use [a-z_] and not be too long.
//
// full name: Any name that describes the plugin nicely.
//
// registerFunc is called once for each function you want to register. Function names
// should be PascalCase. The argument string has this format:
// name:type; or name:type:flag1:flag2....;
// All argument name should be lowercase and only use [a-z_].
// The valid types are int,float,data,clip,frame,func. [] can be appended to allow arrays
// of type to be passed (numbers:int[])
// The available flags are opt, to make an argument optional, empty, which controls whether
// or not empty arrays are accepted and link which will not be explained here.

VS_EXTERNAL_API(void) VapourSynthPluginInit(VSConfigPlugin configFunc, VSRegisterFunction registerFunc, VSPlugin *plugin) {
    configFunc("com.example.invert", "cuinvert", "VapourSynth Invert Example", VAPOURSYNTH_API_VERSION, 1, plugin);
    registerFunc("Filter", "clip:clip;enabled:int:opt;", invertCreate, 0, plugin);
}
