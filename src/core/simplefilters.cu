#include "hip/hip_runtime.h"
// Contains the CUDA implementations of the simplefilters operations.
#include "VapourSynth.h"
#include "VSCuda.h"

///////////////////////
// Merge


//This kernel operates by each thread fetching a stretch of 4 8-bit pixels,
//operating on them, and then sending them back to the destination.
//This is done to achieve coalesced memory accesses, which are crucial for
//high performance in CUDA.
static __global__ void mergeKernel(uint8_t *dstp, const uint8_t *srcp1, const uint8_t *srcp2, const int stride, const int width, const int height, const int weight, const int round, const int MergeShift){
    const int column = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const int row = IMAD(blockDim.y, blockIdx.y, threadIdx.y);

    if (column >= width || row >= height)
        return;

   uint32_t src1_data = ((uint32_t *)srcp1)[(stride / sizeof(uint32_t)) * row + column];
   uint32_t src2_data = ((uint32_t *)srcp2)[(stride / sizeof(uint32_t)) * row + column];
   uint32_t dst_data = 0;

   for (int i = 0; i < sizeof(uint32_t); i++) {
      ((uint8_t *)&dst_data)[i] = ((uint8_t *)&src1_data)[i] + (((((uint8_t *)&src2_data)[i] - ((uint8_t *)&src1_data)[i]) * weight + round) >> MergeShift);
   }

   //dstp[x] = srcp1[x] + (((srcp2[x] - srcp1[x]) * weight + round) >> MergeShift);
   ((uint32_t *)dstp)[(stride / sizeof(uint32_t)) * row + column] = dst_data;
}

VS_EXTERN_C void VS_CC mergeProcessCUDA(uint8_t *dstp, const uint8_t *srcp1, const uint8_t *srcp2, const int stride, const int width, const int height, const int weight, const int round, const int MergeShift) {
   hipDeviceProp_t * deviceProp = VSCUDAGetDefaultDeviceProperties();

   int blockSize = (deviceProp->major < 2) ? 16 : 32;

   dim3 threads(blockSize, blockSize);
   dim3 grid(ceil((float)width / (threads.x * sizeof(uint32_t))), ceil((float)height / threads.y));

   mergeKernel<<<grid, threads>>>(dstp, srcp1, srcp2, stride, width / sizeof(uint32_t), height, weight, round, MergeShift);
}

