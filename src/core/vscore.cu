/*
* Copyright (c) 2012 Fredrik Mellbin, Austin Dworaczyk Wiltshire
*
* This file is part of VapourSynth.
*
* VapourSynth is free software; you can redistribute it and/or
* modify it under the terms of the GNU Lesser General Public
* License as published by the Free Software Foundation; either
* version 2.1 of the License, or (at your option) any later version.
*
* VapourSynth is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
* Lesser General Public License for more details.
*
* You should have received a copy of the GNU Lesser General Public
* License along with VapourSynth; if not, write to the Free Software
* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
*/

// A CUDA specific file for all CUDA-aimed functions of the VSCore.

#include "vscore.h"
#include "VSCuda.h"
#include "VSHelper.h"

//Note: FrameLocation is necessary in order to manage memory correctly in the VSFrameData destructor.
VSFrameData::VSFrameData(int width, int height, int *stride, int bytesPerSample, MemoryUse * mem,
                         FrameLocation fLocation) : mem(mem), frameLocation(fLocation) {
    hipPitchedPtr d_ptr;

    if (fLocation != flGPU) {
        qFatal("Only GPU memory allocation is currently supported by this function. This needs to be fixed.");
    }

    CHECKCUDA(hipMalloc3D(&d_ptr, make_hipExtent(width * bytesPerSample, height, 1)));
    data = (uint8_t *) d_ptr.ptr;
    *stride = d_ptr.pitch;
    size = *stride * height;
    mem->add(size);
}

VSFrameData::VSFrameData(const VSFrameData &d) : QSharedData(d) {
    size = d.size;
    mem = d.mem;
    frameLocation = d.frameLocation;

    if (frameLocation == flLocal) {
        data = vs_aligned_malloc<uint8_t>(size, VSFrame::alignment);
        Q_CHECK_PTR(data);
        memcpy(data, d.data, size);
    } else {
        CHECKCUDA(hipMalloc(&data, size));
        CHECKCUDA(hipMemcpy(data, d.data, size, hipMemcpyDeviceToDevice));
    }

    mem->add(size);
}

VSFrameData::~VSFrameData() {
    if (frameLocation == flLocal)
        vs_aligned_free(data);
    else
        CHECKCUDA(hipFree(data));

    mem->subtract(size);
}

//Transfer video frame data asynchronously using the given cudaStream.
void VSFrameData::transferData(VSFrameData *dst, int dstStride,
                               int srcStride, int width, int height, int bytesPerSample,
                               FrameTransferDirection direction) const {
    hipMemcpyKind transferKind = (direction == ftdCPUtoGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost);

    //We are not using hipMemcpy2DAsync for a reason here.
    //Strangely, it offers no performance benefit, and tests showed a strange glitch error
    //occured on the first few frames of a clip with certain combinations of the Merge filter.
    //No idea why the error occurred, but using hipMemcpy2D (no async) is safe (glitch-free)
    //and offers the same performance.
    CHECKCUDA(hipMemcpy2D(dst->data, dstStride, data, srcStride, width * bytesPerSample, height, transferKind));
}

//Note: future integration can use default parameters to prevent code duplication.
VSFrame::VSFrame(const VSFormat * f, int width, int height, const VSFrame * propSrc, VSCore * core,
                 FrameLocation fLocation) : format(f), width(width), height(height),
                 frameLocation(fLocation) {
    if (!f || width <= 0 || height <= 0)
        qFatal("Invalid new frame");

    if (propSrc)
        properties = propSrc->properties;

    if (frameLocation != flLocal && frameLocation != flGPU)
        qFatal("Invalid frame location. Please use flLocal or flGPU. Specified: %d", frameLocation);

    if (format->numPlanes != 3) {
        stride[1] = 0;
        stride[2] = 0;
    }

    if (frameLocation == flLocal) {
        //Handle CPU implementation.
        //This is a simple copy and paste of the vscore.cpp VSFrame constructor.
        stride[0] = (width * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);

        if (f->numPlanes == 3) {
            int plane23 = ((width >> f->subSamplingW) * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);
            stride[1] = plane23;
            stride[2] = plane23;
        }

        data[0] = new VSFrameData(stride[0] * height, core->memory);
        if (f->numPlanes == 3) {
            int size23 = stride[1] * (height >> f->subSamplingH);
            data[1] = new VSFrameData(size23, core->memory);
            data[2] = new VSFrameData(size23, core->memory);
        }
    } else {
        //Handle GPU implementation.
        for (int plane = 0; plane < f->numPlanes; plane++) {
            int compensatedWidth  = (plane ? width  >> f->subSamplingW : width);
            int compensatedHeight = (plane ? height >> f->subSamplingH : height);

            data[plane] =
                new VSFrameData(compensatedWidth, compensatedHeight, &stride[plane], f->bytesPerSample,
                                core->gpuMemory, frameLocation);
        }
    }
}

VSFrame::VSFrame(const VSFormat *f, int width, int height, const VSFrame * const *planeSrc, const int *plane, const VSFrame *propSrc, VSCore *core, FrameLocation fLocation) : format(f), width(width), height(height), frameLocation(fLocation) {
    if (!f || width <= 0 || height <= 0)
        qFatal("Invalid new frame");

    if (propSrc)
        properties = propSrc->properties;

    if (format->numPlanes != 3) {
        stride[1] = 0;
        stride[2] = 0;
    }

    //Calculate the stride.
    //WARNING: This stride gets over written when allocating on the GPU, in order to meet
    //GPU memory alignment requirements.
    stride[0] = (width * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);

    if (f->numPlanes == 3) {
        int plane23 = ((width >> f->subSamplingW) * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);
        stride[1] = plane23;
        stride[2] = plane23;
    }

    for (int i = 0; i < format->numPlanes; i++) {
        if (planeSrc[i]) {
            if (plane[i] < 0 || plane[i] >= planeSrc[i]->format->numPlanes)
                qFatal("Plane does no exist, error in frame creation");
            if (planeSrc[i]->getHeight(plane[i]) != getHeight(i) || planeSrc[i]->getWidth(plane[i]) != getWidth(i))
                qFatal("Copied plane dimensions do not match, error in frame creation");
            data[i] = planeSrc[i]->data[plane[i]];
            stride[i] = planeSrc[i]->stride[plane[i]];
        } else {
            int compensatedWidth  = (i ? width  >> f->subSamplingW : width);
            int compensatedHeight = (i ? height >> f->subSamplingH : height);

            if (frameLocation == flLocal)
                data[i] = new VSFrameData(stride[i] * compensatedHeight, core->memory);
            else
                data[i] = new VSFrameData(compensatedWidth, compensatedHeight, &stride[i], f->bytesPerSample,
                            core->gpuMemory, frameLocation);
        }
    }
}

void VSFrame::transferFrame(VSFrame &dstFrame, FrameTransferDirection direction) const {
    if(dstFrame.width != width || dstFrame.height != height)
        qFatal("The source frame and destination frame dimensions do not match.");

    //Double check the strides, just to make sure cuda isn't pulling any nasty shit.
    //This can probably safely be removed.
    if(dstFrame.format->numPlanes != format->numPlanes)
        qFatal("The source frame and destination frame do not have the same number of planes.");

    for(int plane = 0; plane < format->numPlanes; plane++) {
        data[plane].data()->transferData(dstFrame.data[plane].data(), dstFrame.stride[plane],
                                                  stride[plane], getWidth(plane),
                                                  getHeight(plane), format->bytesPerSample, direction);
    }
}

PVideoFrame VSCore::newVideoFrame(const VSFormat *f, int width, int height, const VSFrame *propSrc, FrameLocation fLocation) {
    return PVideoFrame(new VSFrame(f, width, height, propSrc, this, fLocation));
}

PVideoFrame VSCore::newVideoFrame(const VSFormat *f, int width, int height, const VSFrame * const *planeSrc, const int *planes, const VSFrame *propSrc, FrameLocation fLocation) {
    return PVideoFrame(new VSFrame(f, width, height, planeSrc, planes, propSrc, this, fLocation));
}

void VSCore::transferVideoFrame(const PVideoFrame &srcf, PVideoFrame &dstf, FrameTransferDirection direction){
    srcf->transferFrame(*dstf.data(), direction);
}

VSGPUManager *VSCore::getGPUManager() const {
    return gpuManager;
}