/*
* Copyright (c) 2012 Fredrik Mellbin, Austin Dworaczyk Wiltshire
*
* This file is part of VapourSynth.
*
* VapourSynth is free software; you can redistribute it and/or
* modify it under the terms of the GNU Lesser General Public
* License as published by the Free Software Foundation; either
* version 2.1 of the License, or (at your option) any later version.
*
* VapourSynth is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
* Lesser General Public License for more details.
*
* You should have received a copy of the GNU Lesser General Public
* License along with VapourSynth; if not, write to the Free Software
* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
*/

// A CUDA specific file for all CUDA-aimed functions of the VSCore.

#include "vscore.h"
#include "VSCuda.h"
#include "VSHelper.h"

//Note: FrameLocation is necessary in order to manage memory correctly in the VSFrameData destructor.
VSFrameData::VSFrameData(int width, int height, int *stride, int bytesPerSample, MemoryUse * mem,
                         FrameLocation fLocation, const VSCUDAStream *stream_in) : mem(mem), frameLocation(fLocation) {
    hipPitchedPtr d_ptr;

    if (fLocation != flGPU) {
        qFatal("Only GPU memory allocation is currently supported by this function. This needs to be fixed.");
    }

    CHECKCUDA(hipMalloc3D(&d_ptr, make_hipExtent(width * bytesPerSample, height, 1)));
    data = (uint8_t *) d_ptr.ptr;
    *stride = d_ptr.pitch;
    stream = stream_in;
    size = *stride * height;
    mem->add(size);
}

VSFrameData::VSFrameData(const VSFrameData &d) : QSharedData(d) {
    size = d.size;
    mem = d.mem;
    frameLocation = d.frameLocation;
    stream = d.stream;

    if (frameLocation == flLocal) {
        data = vs_aligned_malloc<uint8_t>(size, VSFrame::alignment);
        Q_CHECK_PTR(data);
        memcpy(data, d.data, size);
    } else {
        CHECKCUDA(hipMalloc(&data, size));
        CHECKCUDA(hipMemcpyAsync(data, d.data, size, hipMemcpyDeviceToDevice, stream->stream));
    }

    mem->add(size);
}

VSFrameData::~VSFrameData() {
    if (frameLocation == flLocal)
        vs_aligned_free(data);
    else
        CHECKCUDA(hipFree(data));

    mem->subtract(size);
}

//Transfer video frame data asynchronously using the given cudaStream.
void VSFrameData::transferData(VSFrameData *dst, int dstStride,
                               int srcStride, int width, int height, int bytesPerSample,
                               FrameTransferDirection direction) const {
    hipMemcpyKind transferKind = (direction == ftdCPUtoGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost);
    const VSCUDAStream *newStream;

    if (direction == ftdCPUtoGPU)
        newStream = dst->stream;
    else
        newStream = stream;

    //hipEvent_t sync;
    //CHECKCUDA(hipEventCreate(&sync, hipEventDisableTiming | hipEventBlockingSync));
    CHECKCUDA(hipMemcpy2DAsync(dst->data, dstStride, data, srcStride, width * bytesPerSample, height, transferKind, newStream->stream));
    //CHECKCUDA(hipEventRecord(sync, newStream->stream));
    //CHECKCUDA(hipEventSynchronize(sync));
    //CHECKCUDA(hipEventDestroy(sync));
}

//Note: future integration can use default parameters to prevent code duplication.
VSFrame::VSFrame(const VSFormat * f, int width, int height, const VSFrame * propSrc, VSCore * core,
                 FrameLocation fLocation, const VSCUDAStream **streams) : format(f), width(width), height(height),
                 frameLocation(fLocation) {
    if (!f || width <= 0 || height <= 0)
        qFatal("Invalid new frame");

    if (propSrc)
        properties = propSrc->properties;

    if (frameLocation != flLocal && frameLocation != flGPU)
        qFatal("Invalid frame location. Please use flLocal or flGPU. Specified: %d", frameLocation);

    if (format->numPlanes != 3) {
        stride[1] = 0;
        stride[2] = 0;
    }

    if (frameLocation == flLocal) {
        //Handle CPU implementation.
        //This is a simple copy and paste of the vscore.cpp VSFrame constructor.
        stride[0] = (width * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);

        if (f->numPlanes == 3) {
            int plane23 = ((width >> f->subSamplingW) * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);
            stride[1] = plane23;
            stride[2] = plane23;
        }

        data[0] = new VSFrameData(stride[0] * height, core->memory);
        if (f->numPlanes == 3) {
            int size23 = stride[1] * (height >> f->subSamplingH);
            data[1] = new VSFrameData(size23, core->memory);
            data[2] = new VSFrameData(size23, core->memory);
        }
    } else {
        //Handle GPU implementation.
        for (int plane = 0; plane < f->numPlanes; plane++) {
            int compensatedWidth  = (plane ? width  >> f->subSamplingW : width);
            int compensatedHeight = (plane ? height >> f->subSamplingH : height);

            data[plane] =
                new VSFrameData(compensatedWidth, compensatedHeight, &stride[plane], f->bytesPerSample,
                                core->gpuMemory, frameLocation, streams[plane]);
        }
    }
}

VSFrame::VSFrame(const VSFormat *f, int width, int height, const VSFrame * const *planeSrc, const int *plane, const VSFrame *propSrc, VSCore *core, FrameLocation fLocation, const VSCUDAStream **streams) : format(f), width(width), height(height), frameLocation(fLocation) {
    if (!f || width <= 0 || height <= 0)
        qFatal("Invalid new frame");

    if (propSrc)
        properties = propSrc->properties;

    if (format->numPlanes != 3) {
        stride[1] = 0;
        stride[2] = 0;
    }

    //Calculate the stride.
    //WARNING: This stride gets over written when allocating on the GPU, in order to meet
    //GPU memory alignment requirements.
    stride[0] = (width * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);

    if (f->numPlanes == 3) {
        int plane23 = ((width >> f->subSamplingW) * (f->bytesPerSample) + (alignment - 1)) & ~(alignment - 1);
        stride[1] = plane23;
        stride[2] = plane23;
    }

    for (int i = 0; i < format->numPlanes; i++) {
        if (planeSrc[i]) {
            if (plane[i] < 0 || plane[i] >= planeSrc[i]->format->numPlanes)
                qFatal("Plane does no exist, error in frame creation");
            if (planeSrc[i]->getHeight(plane[i]) != getHeight(i) || planeSrc[i]->getWidth(plane[i]) != getWidth(i))
                qFatal("Copied plane dimensions do not match, error in frame creation");
            data[i] = planeSrc[i]->data[plane[i]];
            stride[i] = planeSrc[i]->stride[plane[i]];
        } else {
            int compensatedWidth  = (i ? width  >> f->subSamplingW : width);
            int compensatedHeight = (i ? height >> f->subSamplingH : height);

            if (frameLocation == flLocal)
                data[i] = new VSFrameData(stride[i] * compensatedHeight, core->memory);
            else
                data[i] = new VSFrameData(compensatedWidth, compensatedHeight, &stride[i], f->bytesPerSample,
                            core->gpuMemory, frameLocation, streams[i]);
        }
    }
}

void VSFrame::transferFrame(VSFrame &dstFrame, FrameTransferDirection direction) const {
    if(dstFrame.width != width || dstFrame.height != height)
        qFatal("The source frame and destination frame dimensions do not match.");

    if(dstFrame.format->numPlanes != format->numPlanes)
        qFatal("The source frame and destination frame do not have the same number of planes.");

    for(int plane = 0; plane < format->numPlanes; plane++) {
        data[plane].data()->transferData(dstFrame.data[plane].data(), dstFrame.stride[plane],
                                                  stride[plane], getWidth(plane),
                                                  getHeight(plane), format->bytesPerSample, direction);
    }
}

const VSCUDAStream *VSFrame::getStream(int plane) const {
    if (plane < 0 || plane >= format->numPlanes)
        qFatal("Invalid plane requested");

    switch (plane) {
    case 0:
        return data[0].constData()->stream;
    case 1:
        return data[1].constData()->stream;
    case 2:
        return data[2].constData()->stream;
    default:
        return NULL;
    }
}

PVideoFrame VSCore::newVideoFrame(const VSFormat *f, int width, int height, const VSFrame *propSrc, FrameLocation fLocation) {
    const VSCUDAStream *streams[3];
    for(int plane = 0; plane < f->numPlanes; plane++) {
        streams[plane] = gpuManager->getStreamAtIndex(gpuManager->getNextStreamIndex());
    }
    return PVideoFrame(new VSFrame(f, width, height, propSrc, this, fLocation, streams));
}

PVideoFrame VSCore::newVideoFrame(const VSFormat *f, int width, int height, const VSFrame * const *planeSrc, const int *planes, const VSFrame *propSrc, FrameLocation fLocation) {
    const VSCUDAStream *streams[3];

    //Only retrive new streams if we don't have a prior source.
    for(int plane = 0; plane < f->numPlanes; plane++) {
        if(!planeSrc[plane])
            streams[plane] = gpuManager->getStreamAtIndex(gpuManager->getNextStreamIndex());
        else
            streams[plane] = NULL;
    }
    return PVideoFrame(new VSFrame(f, width, height, planeSrc, planes, propSrc, this, fLocation, streams));
}

void VSCore::transferVideoFrame(const PVideoFrame &srcf, PVideoFrame &dstf, FrameTransferDirection direction){
    srcf->transferFrame(*dstf.data(), direction);
}

VSGPUManager *VSCore::getGPUManager() const {
    return gpuManager;
}
